
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor firstParallel so that it can run on the GPU.
 */

__global__ void firstParallel()
{
  printf("This should be running in parallel.\n");
}

int main()
{
  
  // run firstParallel in 4 thread blocks each containing 5 threads
  firstParallel<<<4, 5>>>();

  // wait until 
  hipDeviceSynchronize();

}
