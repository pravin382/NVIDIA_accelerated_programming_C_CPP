
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void firstParallel()
{
  printf("This should be running in parallel.\n");
}

int main()
{
  
  // run firstParallel in 4 thread blocks each containing 5 threads
  firstParallel<<<4, 5>>>();

  // wait until gpu operations are complete
  hipDeviceSynchronize();

}
