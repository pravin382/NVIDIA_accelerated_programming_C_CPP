
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop() //no need of N
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    printf("This is iteration number %d\n", i);

}

int main()
{


  int N = 10;
  int blocks = 2;
  loop<<<blocks,N/blocks>>>();

  hipDeviceSynchronize();
}
