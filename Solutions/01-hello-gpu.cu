
#include <hip/hip_runtime.h>
#include <stdio.h>

// no need to change this
void helloCPU()
{
  printf("Hello from the CPU.\n");
}


// add __GLOBAL__ so that the function runs from gpu
__global__ void helloGPU()
{
  printf("Hello from the GPU.\n");
}

int main()
{

  // calling the  GPU function
  helloGPU<<<1, 1>>>();
  hipDeviceSynchronize(); // finish GPU first and then move to next

  helloCPU();

  helloGPU<<<1, 1>>>();
  // synchronize CPU and GPU operations
  hipDeviceSynchronize();

} 


