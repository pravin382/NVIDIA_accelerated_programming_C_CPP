
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printSuccessForCorrectExecutionConfiguration()
{

  if(threadIdx.x == 3 && blockIdx.x == 4)
  {
    printf("Success!\n");
  } else {
    //printf("Failure. Update the execution configuration as necessary.\n");
  }
}

int main()
{
  
  // altering such that we get success once
  printSuccessForCorrectExecutionConfiguration<<<5, 4>>>();

  hipDeviceSynchronize();
}
